
#include <hip/hip_runtime.h>
#include <iostream>
#include <pthread.h>

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


const int UNPROCESSED = 1;
const int PROCESSED = 2;
const int DONE = 3;
const int RUNNING= 0;
const int NUM_OF_DATA = 10;
const int NUM_OF_BLOCKS = 1;
const int THREADS_PER_BLOCK = 1;

//int data_states[NUM_OF_DATA];
hipStream_t hip_stream[2];
volatile int* process_state;
volatile int* d_process_state;
volatile int* d_copier_state;
int* d_data_state;
int* h_data_states;
int* h_copier_state;
hipError_t cuda_status;

using namespace std;

void* copy_data(void* arg){
   int i=0;
   hipSetDevice(0);
   cout << "in copy_data" << endl;
   while(i < NUM_OF_DATA){
      if (*process_state != UNPROCESSED){
        cout << "Now copy data " << i << " with state = " << h_data_states[i] <<  endl;
         *process_state = UNPROCESSED;
         hipMemcpyAsync(d_data_state, &(h_data_states[i]), sizeof(int), hipMemcpyHostToDevice, hip_stream[0]);
         hipStreamSynchronize(hip_stream[0]);
         cudaCheckErrors("thread hipMemcpyAsync fail");
         //*process_state = UNPROCESSED;
         i++;
         }
      }
   *h_copier_state = DONE;
   hipMemcpyAsync((void *)d_copier_state, h_copier_state, sizeof(int), hipMemcpyHostToDevice, hip_stream[0]);
   cudaCheckErrors("thread hipMemcpyAsync 2 fail");
  cout << "Thread finished" << endl;
   return NULL;
}

__global__ void process_data(volatile int* data_state, volatile int* process_state, volatile int* copier_state){
   int i = 0;
   printf("In process data: i = %d\n", i);
   while(*copier_state != DONE){
      printf(" i = %d, copier_state = %d, data_state = %d\n", i, *copier_state, *data_state);
      if(*data_state == UNPROCESSED){
        printf("now processing data %d\n", i);
        i++;
        // process data here, skipped
        *data_state = PROCESSED;
        __threadfence_system();
        *process_state = PROCESSED;
        __threadfence_system();
      }
   }
  printf("process_data is done\n");
}

int main(int argc, char **argv){
  int i;
  float elapsed_time;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  hipEventRecord(stop, 0);

  hipSetDevice(0);
  hipSetDeviceFlags(hipDeviceMapHost);
  hipHostMalloc((void**) &process_state, NUM_OF_BLOCKS*sizeof(int),  hipHostMallocMapped, hipHostMallocDefault);
  cudaCheckErrors("hipHostMalloc 1 fail");
  hipHostGetDevicePointer((int**) &d_process_state, (int*) process_state, 0);
  hipMalloc((void**) &d_copier_state, sizeof(int));
  cudaCheckErrors("hipMalloc 1 fail");
  hipMemset((void*)d_copier_state, RUNNING, sizeof(int));
  hipHostMalloc((void**) &h_copier_state, sizeof(int), 0, hipHostMallocDefault);
  cudaCheckErrors("hipHostMalloc 3 fail");
  *h_copier_state = RUNNING;
  hipHostMalloc((void**) &h_data_states, NUM_OF_DATA*sizeof(int), 0, hipHostMallocDefault);
  cudaCheckErrors("hipHostMalloc 2 fail");
  for(i = 0; i < NUM_OF_DATA; i++){
     h_data_states[i] = UNPROCESSED;
  }
  hipMalloc((void**) &d_data_state, sizeof(int));
  cudaCheckErrors("hipMalloc 2 fail");
  hipMemcpy((void*)d_data_state, &(h_data_states[0]), sizeof(int), hipMemcpyHostToDevice);

  hipStreamCreate(&hip_stream[0]);
  hipStreamCreate(&hip_stream[1]);
  pthread_t thread;
  int thread_state = pthread_create(&thread, NULL, &copy_data, NULL);
  if(thread_state){
     cout << "Error: unable to create thread (produce_instances), "<< thread_state << endl;
     exit(-1);
  }


  cout << "Starting kernel" << endl;
  process_data<<<NUM_OF_BLOCKS, THREADS_PER_BLOCK, 0, hip_stream[1]>>>(d_data_state, d_process_state, d_copier_state);


  hipDeviceSynchronize();

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  cout << "Elapsed time: " << elapsed_time << " ms" << endl;
  return 0;
}